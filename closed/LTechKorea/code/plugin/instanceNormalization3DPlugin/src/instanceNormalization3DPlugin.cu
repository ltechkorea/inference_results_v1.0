#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION. All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <stdexcept>

#include "instanceNormalization3DPlugin.h"

using namespace nvinfer1;
using nvinfer1::plugin::InstanceNormalization3DPlugin;
using nvinfer1::plugin::InstanceNormalization3DPluginCreator;

#define CHECK_CUDA(call)                                                                                               \
    do                                                                                                                 \
    {                                                                                                                  \
        hipError_t status = call;                                                                                     \
        if (status != hipSuccess)                                                                                     \
        {                                                                                                              \
            return status;                                                                                             \
        }                                                                                                              \
    } while (0)

#define CHECK_CUDNN(call)                                                                                              \
    do                                                                                                                 \
    {                                                                                                                  \
        hipdnnStatus_t status = call;                                                                                   \
        if (status != HIPDNN_STATUS_SUCCESS)                                                                            \
        {                                                                                                              \
            return status;                                                                                             \
        }                                                                                                              \
    } while (0)

template<typename T, int THREADS_PER_CTA>
__global__ __launch_bounds__(THREADS_PER_CTA)
void in3d_relu_activation(T* __restrict dst, T* __restrict src, float alpha, int count)
{
    int idx = blockIdx.x * THREADS_PER_CTA + threadIdx.x;
    if (idx >= count) return;

    float val = src[idx];
    dst[idx] = (val < 0.f)? val * alpha : val;
}

// This is derived from: https://fgiesen.wordpress.com/2012/03/28/half-to-float-done-quic/
inline float half_to_float_fast(unsigned short value)
{
    union F32
    {
        unsigned int u;
        float f;
    };
    static const F32 magic = {(254 - 15) << 23};
    static const F32 was_infnan = {(127 + 16) << 23};
    F32 result;
    result.u = (value & 0x7fff) << 13; // exponent/mantissa bits
    result.f *= magic.f;               // exponent adjust
    if (result.f >= was_infnan.f)
    { // make sure Inf/NaN survive
        result.u |= 255 << 23;
    }
    result.u |= (value & 0x8000) << 16; // sign bit
    return result.f;
}

hipdnnStatus_t convert_trt2cudnn_dtype(nvinfer1::DataType trt_dtype, hipdnnDataType_t* cudnn_dtype)
{
    switch (trt_dtype)
    {
    case nvinfer1::DataType::kFLOAT: *cudnn_dtype = HIPDNN_DATA_FLOAT; break;
    case nvinfer1::DataType::kHALF: *cudnn_dtype = HIPDNN_DATA_HALF; break;
    default: return HIPDNN_STATUS_BAD_PARAM;
    }
    return HIPDNN_STATUS_SUCCESS;
}

namespace {
    const char* INSTNORM3D_PLUGIN_VERSION{"1"};
    const char* INSTNORM3D_PLUGIN_NAME{"INSTNORM3D_TRT"};
}

REGISTER_TENSORRT_PLUGIN(InstanceNormalization3DPluginCreator);

PluginFieldCollection InstanceNormalization3DPluginCreator::mFC{};
std::vector<PluginField> InstanceNormalization3DPluginCreator::mPluginAttributes;

InstanceNormalization3DPlugin::InstanceNormalization3DPlugin(
    float epsilon, const std::vector<float>& scale, const std::vector<float>& bias, int relu, float alpha)
    : _epsilon(epsilon)
    , _nchan(scale.size())
    , _h_scale(scale)
    , _h_bias(bias)
    , _relu(relu)
    , _alpha(alpha)
    , _in_scale(-1.f)
    , _out_scale(-1.f)
{
    ASSERT(scale.size() == bias.size());
}

InstanceNormalization3DPlugin::InstanceNormalization3DPlugin(
    float epsilon, nvinfer1::Weights const& scale, nvinfer1::Weights const& bias, int relu, float alpha)
    : _epsilon(epsilon)
    , _nchan(scale.count)
    , _relu(relu)
    , _alpha(alpha)
    , _in_scale(-1.f)
    , _out_scale(-1.f)
{
    ASSERT(scale.count == bias.count);
    if (scale.type == nvinfer1::DataType::kFLOAT)
    {
        _h_scale.assign((float*) scale.values, (float*) scale.values + scale.count);
    }
    else if (scale.type == nvinfer1::DataType::kHALF)
    {
        _h_scale.reserve(_nchan);
        for (int c = 0; c < _nchan; ++c)
        {
            unsigned short value = ((unsigned short*) scale.values)[c];
            _h_scale.push_back(half_to_float_fast(value));
        }
    }
    else
    {
        throw std::runtime_error("Unsupported scale dtype");
    }
    if (bias.type == nvinfer1::DataType::kFLOAT)
    {
        _h_bias.assign((float*) bias.values, (float*) bias.values + bias.count);
    }
    else if (bias.type == nvinfer1::DataType::kHALF)
    {
        _h_bias.reserve(_nchan);
        for (int c = 0; c < _nchan; ++c)
        {
            unsigned short value = ((unsigned short*) bias.values)[c];
            _h_bias.push_back(half_to_float_fast(value));
        }
    }
    else
    {
        throw std::runtime_error("Unsupported bias dtype");
    }
}

InstanceNormalization3DPlugin::InstanceNormalization3DPlugin(void const* serialData, size_t serialLength)
{
    deserialize_value(&serialData, &serialLength, &_epsilon);
    deserialize_value(&serialData, &serialLength, &_nchan);
    deserialize_value(&serialData, &serialLength, &_h_scale);
    deserialize_value(&serialData, &serialLength, &_h_bias);
    deserialize_value(&serialData, &serialLength, &_relu);
    deserialize_value(&serialData, &serialLength, &_alpha);
    deserialize_value(&serialData, &serialLength, &_in_scale);
    deserialize_value(&serialData, &serialLength, &_out_scale);
}

InstanceNormalization3DPlugin::~InstanceNormalization3DPlugin()
{
    terminate();
}

// InstanceNormalization3DPlugin returns one output.
int InstanceNormalization3DPlugin::getNbOutputs() const
{
    return 1;
}

DimsExprs InstanceNormalization3DPlugin::getOutputDimensions(
    int outputIndex, const nvinfer1::DimsExprs* inputs, int nbInputs, nvinfer1::IExprBuilder& exprBuilder)
{
    nvinfer1::DimsExprs output(inputs[0]);
    return output;
}

int InstanceNormalization3DPlugin::initialize()
{
    if (!initialized)
    {
        CHECK_CUDNN(hipdnnCreate(&_cudnn_handle));

        CHECK_CUDNN(hipdnnCreateTensorDescriptor(&_b_desc));
        CHECK_CUDNN(hipdnnCreateTensorDescriptor(&_x_desc));
        CHECK_CUDNN(hipdnnCreateTensorDescriptor(&_y_desc));

        // NDHWC path
        // Device info.
        int device;
        CHECK_CUDA(hipGetDevice(&device));
        hipDeviceProp_t props;
        CHECK_CUDA(hipGetDeviceProperties(&props, device));

        _context.sm_count = props.multiProcessorCount;
        _context.sm_shared_size = props.sharedMemPerMultiprocessor;
        _context.sm_version = props.major * 100 + props.minor * 10;

        memset(&_params, 0, sizeof(_params));

        CHECK_CUDA(hipMalloc(&_d_scale, _nchan*sizeof(float)));
        CHECK_CUDA(hipMalloc(&_d_bias, _nchan*sizeof(float)));
        CHECK_CUDA(hipMemcpy(_d_scale, &_h_scale[0], _nchan*sizeof(float), hipMemcpyHostToDevice));
        CHECK_CUDA(hipMemcpy(_d_bias, &_h_bias[0], _nchan*sizeof(float), hipMemcpyHostToDevice));
    }
    initialized = true;
    return 0;
}

void InstanceNormalization3DPlugin::terminate()
{
    if (initialized)
    {
        hipdnnDestroyTensorDescriptor(_y_desc);
        hipdnnDestroyTensorDescriptor(_x_desc);
        hipdnnDestroyTensorDescriptor(_b_desc);

        hipdnnDestroy(_cudnn_handle);

        hipFree(_d_bias);
        hipFree(_d_scale);
    }
    initialized = false;
    return;
}

size_t InstanceNormalization3DPlugin::getWorkspaceSize(const nvinfer1::PluginTensorDesc* inputs, int nbInputs, const nvinfer1::PluginTensorDesc* outputs, int nbOutputs) const 
{ 
    if (inputs[0].format == nvinfer1::PluginFormat::kLINEAR)
    {
        nvinfer1::Dims     input_dims = inputs[0].dims;

        int n = input_dims.d[0];
        int c = input_dims.d[1];

        size_t nchan_bytes = c * sizeof(float);
        size_t scale_size = n * nchan_bytes;
        size_t bias_size  = n * nchan_bytes;

        size_t total_wss = scale_size + bias_size;
        
        return total_wss;
    }
    else if (inputs[0].format == nvinfer1::PluginFormat::kDHWC8 ||
             inputs[0].format == nvinfer1::PluginFormat::kCDHW32)
    {
        int input_data_type = (inputs[0].type == nvinfer1::DataType::kHALF) ? 1 : 2;
        int output_data_type = (outputs[0].type == nvinfer1::DataType::kHALF) ? 1 : 2;
        nvinfer1::Dims     input_dims = inputs[0].dims;

        int n = input_dims.d[0];
        int c = input_dims.d[1];
        int d = input_dims.d[2];
        int h = input_dims.d[3];
        int w = input_dims.d[4];
        
        InstanceNormFwdParams params;
        // only these parameters are required for workspace computation
        params.nhw = d*h*w;
        params.c = c;
        params.n = n;
        // Reserve memory for the workspaces.
        size_t size_sums, size_counts, size_retired_ctas;
        instance_norm_buffer_sizes_dispatch(_context, params, size_sums, size_counts, size_retired_ctas,
                                            input_data_type, output_data_type);
        size_t size_nc = n*c*sizeof(float);
        size_nc = ((size_nc + 256 - 1) / 256) * 256;
        return size_sums + size_counts + size_retired_ctas + 4 * size_nc;
    }
    else
    {
        ASSERT(0);
    }
}


int InstanceNormalization3DPlugin::enqueue(const nvinfer1::PluginTensorDesc* inputDesc,
    const nvinfer1::PluginTensorDesc* outputDesc, const void* const* inputs, void* const* outputs, void* workspace,
    hipStream_t stream)
{
    ASSERT(initialized);

    if (inputDesc[0].format == nvinfer1::PluginFormat::kLINEAR)
    {
        CHECK_CUDNN(hipdnnSetStream(_cudnn_handle, stream));
        nvinfer1::Dims input_dims = inputDesc[0].dims;
        int n = input_dims.d[0];
        int c = input_dims.d[1];
        int d = input_dims.d[2];
        int h = input_dims.d[3];
        int w = input_dims.d[4];
        size_t nchan_bytes = c * sizeof(float);

        // Note: We repeat the data for each batch entry so that we can do the full
        //       computation in a single CUDNN call in enqueue().
        float* _d_array = (float*)workspace;
        float* d_scale = &_d_array[0];
        float* d_bias  = &_d_array[n*c];
        for (int i = 0; i < n; ++i)
        {
            CHECK_CUDA(hipMemcpyAsync(d_scale + i * c, _d_scale, nchan_bytes, hipMemcpyDeviceToDevice, stream));
            CHECK_CUDA(hipMemcpyAsync(d_bias + i * c, _d_bias, nchan_bytes, hipMemcpyDeviceToDevice, stream));
        }

        int nc_dimA[] = {1, n*c, 1, 1, 1};
        int nc_strideA[] = {nc_dimA[1]*nc_dimA[2]*nc_dimA[3]*nc_dimA[4],\
                            nc_dimA[2]*nc_dimA[3]*nc_dimA[4],\
                            nc_dimA[3]*nc_dimA[4],\
                            nc_dimA[4],\
                            1};
        int img_dimA[] = {1, n*c, d, h, w};
        int img_strideA[] = {img_dimA[1]*img_dimA[2]*img_dimA[3]*img_dimA[4],\
                            img_dimA[2]*img_dimA[3]*img_dimA[4],\
                            img_dimA[3]*img_dimA[4],\
                            img_dimA[4],\
                            1};

        CHECK_CUDNN(hipdnnSetTensorNdDescriptor(_b_desc, HIPDNN_DATA_FLOAT, 5, nc_dimA, nc_strideA));
        hipdnnDataType_t cudnn_dtype;
        CHECK_CUDNN(convert_trt2cudnn_dtype(inputDesc[0].type, &cudnn_dtype));
        CHECK_CUDNN(hipdnnSetTensorNdDescriptor(_x_desc, cudnn_dtype, 5, img_dimA, img_strideA));
        CHECK_CUDNN(hipdnnSetTensorNdDescriptor(_y_desc, cudnn_dtype, 5, img_dimA, img_strideA));
        float alpha = 1;
        float beta = 0;

        //hipStreamSynchronize(stream);
        void const* x_ptr = inputs[0];
        void* y_ptr = outputs[0];
        // Note: Use of HIPDNN_BATCHNORM_SPATIAL_PERSISTENT can cause numerical
        //       overflows (NaNs) for fp32 data in some circumstances. The lower-
        //       performance HIPDNN_BATCHNORM_SPATIAL should be used if this is not
        //       acceptable.
        CHECK_CUDNN(hipdnnBatchNormalizationForwardTraining(_cudnn_handle, HIPDNN_BATCHNORM_SPATIAL_PERSISTENT, &alpha, &beta,
            _x_desc, x_ptr, _y_desc, y_ptr, _b_desc, d_scale, d_bias, 1., nullptr, nullptr, _epsilon, nullptr, nullptr));

        if (_relu > 0)
        {
            int count = n * c * d * h * w;
            const int BLOCK_SZ = 256;
            if (inputDesc[0].type == nvinfer1::DataType::kFLOAT)
            {
                in3d_relu_activation<float, BLOCK_SZ><<<(count + BLOCK_SZ - 1) / BLOCK_SZ, BLOCK_SZ, 0, stream>>>((float *)y_ptr, (float *)y_ptr, _alpha, count);
            } else if (inputDesc[0].type == nvinfer1::DataType::kHALF)
            {
                in3d_relu_activation<__half, BLOCK_SZ><<<(count + BLOCK_SZ - 1) / BLOCK_SZ, BLOCK_SZ, 0, stream>>>((__half *)y_ptr, (__half *)y_ptr, _alpha, count);
            }
            else
            {
                ASSERT(0);
            }
        }
    }
    else if (inputDesc[0].format == nvinfer1::PluginFormat::kDHWC8 ||
             inputDesc[0].format == nvinfer1::PluginFormat::kCDHW32)
    {
        int input_data_type = (inputDesc[0].type == nvinfer1::DataType::kHALF) ? 1 : 2;
        int output_data_type = (outputDesc[0].type == nvinfer1::DataType::kHALF) ? 1 : 2;

        nvinfer1::Dims input_dims = inputDesc[0].dims;
        int n = input_dims.d[0];
        int c = input_dims.d[1];
        int d = input_dims.d[2];
        int h = input_dims.d[3];
        int w = input_dims.d[4];

        _params.nhw = d*h*w;
        _params.c = c;
        _params.n = n;

        size_t size_sums, size_counts, size_retired_ctas;
        instance_norm_buffer_sizes_dispatch(_context, _params, size_sums, size_counts, size_retired_ctas,
                                            input_data_type, output_data_type);

        size_t size_nc = n*c*sizeof(float);
        size_nc = ((size_nc + 256 - 1) / 256) * 256;

        char* d_buf = reinterpret_cast<char *>(workspace);

        _params.gmem_sums = reinterpret_cast<GMEM_SUMS_TYPE *>(d_buf); d_buf += size_sums;
        _params.gmem_counts = reinterpret_cast<int *>(d_buf); d_buf += size_counts;
        _params.gmem_retired_ctas = reinterpret_cast<int *>(d_buf); d_buf += size_retired_ctas;
        _params.gmem_running_mean = reinterpret_cast<float *>(d_buf); d_buf += size_nc;
        _params.gmem_running_var = reinterpret_cast<float *>(d_buf); d_buf += size_nc;
        _params.gmem_saved_mean = reinterpret_cast<float *>(d_buf); d_buf += size_nc;
        _params.gmem_saved_var = reinterpret_cast<float *>(d_buf); d_buf += size_nc;

        _params.gmem_src = const_cast<void *>(inputs[0]);
        _params.gmem_dst = outputs[0];
        _params.gmem_bias = _d_bias;
        _params.gmem_scale = _d_scale;

        _params.var_eps = _epsilon;
        _params.exp_avg_factor = 1.f; //(float)exp_avg_factor;
        _params.use_relu = _relu; //use_relu;
        _params.relu_alpha = _alpha; //relu_alpha;

        _params.in_scale = _in_scale;
        _params.out_scale = 1.f / _out_scale;

        int loop = instance_norm_fwd_dispatch(_context, _params, stream, input_data_type, output_data_type);
    }
    else
    {
        ASSERT(false && "Unexpected input format");
    }

    return 0;
}

size_t InstanceNormalization3DPlugin::getSerializationSize() const
{
    return (serialized_size(_epsilon) +
            serialized_size(_nchan) +
            serialized_size(_h_scale) +
            serialized_size(_h_bias)) +
            serialized_size(_relu) +
            serialized_size(_alpha) +
            serialized_size(_in_scale) +
            serialized_size(_out_scale);
}

void InstanceNormalization3DPlugin::serialize(void *buffer) const
{
    serialize_value(&buffer, _epsilon);
    serialize_value(&buffer, _nchan);
    serialize_value(&buffer, _h_scale);
    serialize_value(&buffer, _h_bias);
    serialize_value(&buffer, _relu);
    serialize_value(&buffer, _alpha);
    serialize_value(&buffer, _in_scale);
    serialize_value(&buffer, _out_scale);
}

bool InstanceNormalization3DPlugin::supportsFormatCombination(
    int pos, const nvinfer1::PluginTensorDesc* inOut, int nbInputs, int nbOutputs)
{
    ASSERT(inOut && pos < (nbInputs + nbOutputs));

    bool support_fp32_linear = (inOut[pos].type == nvinfer1::DataType::kFLOAT
            && inOut[pos].format == nvinfer1::PluginFormat::kLINEAR
            && inOut[pos].type == inOut[0].type
            && inOut[pos].format == inOut[0].format);

    bool support_fp16_dhwc8 = (inOut[pos].type == nvinfer1::DataType::kHALF
        && inOut[pos].format == nvinfer1::PluginFormat::kDHWC8
        && inOut[pos].type == inOut[0].type
        && inOut[pos].format == inOut[0].format);

    bool support_int8_cdhw32 = (inOut[pos].type == nvinfer1::DataType::kINT8
        && inOut[pos].format == nvinfer1::PluginFormat::kCDHW32
        && inOut[pos].type == inOut[0].type
        && inOut[pos].format == inOut[0].format);

    ASSERT(pos == 0 || pos == 1);

    return support_fp32_linear || support_fp16_dhwc8 || support_int8_cdhw32;
}

const char* InstanceNormalization3DPlugin::getPluginType() const
{
    return INSTNORM3D_PLUGIN_NAME;
}

const char* InstanceNormalization3DPlugin::getPluginVersion() const
{
    return INSTNORM3D_PLUGIN_VERSION;
}

void InstanceNormalization3DPlugin::destroy()
{ 
    delete this;
}

IPluginV2DynamicExt* InstanceNormalization3DPlugin::clone() const
{ 
    auto plugin = new InstanceNormalization3DPlugin{_epsilon, _h_scale, _h_bias, _relu, _alpha};
    plugin->setPluginNamespace(mPluginNamespace);
    plugin->initialize();
    return plugin;
}

// Set plugin namespace
void InstanceNormalization3DPlugin::setPluginNamespace(const char* pluginNamespace)
{
    mPluginNamespace = pluginNamespace;
}

const char* InstanceNormalization3DPlugin::getPluginNamespace() const
{
    return mPluginNamespace;
}

nvinfer1::DataType InstanceNormalization3DPlugin::getOutputDataType(
    int index, const nvinfer1::DataType* inputTypes, int nbInputs) const
{
    ASSERT(inputTypes && nbInputs > 0 && index == 0);

    return nvinfer1::DataType::kFLOAT;
}

void InstanceNormalization3DPlugin::configurePlugin(const nvinfer1::DynamicPluginTensorDesc* in, int nbInputs,
    const nvinfer1::DynamicPluginTensorDesc* out, int nbOutputs)
{
    _in_scale = in[0].desc.scale;
    _out_scale = out[0].desc.scale;
}

// InstanceNormalization3DPluginCreator methods
InstanceNormalization3DPluginCreator::InstanceNormalization3DPluginCreator()
{
    mPluginAttributes.emplace_back(PluginField("epsilon", nullptr, PluginFieldType::kFLOAT32, 1));
    mPluginAttributes.emplace_back(PluginField("scales", nullptr, PluginFieldType::kFLOAT32, 1));
    mPluginAttributes.emplace_back(PluginField("bias", nullptr, PluginFieldType::kFLOAT32, 1));
    mPluginAttributes.emplace_back(PluginField("relu", nullptr, PluginFieldType::kINT32, 1));
    mPluginAttributes.emplace_back(PluginField("alpha", nullptr, PluginFieldType::kFLOAT32, 1));

    mFC.nbFields = mPluginAttributes.size();
    mFC.fields = mPluginAttributes.data();
}

const char* InstanceNormalization3DPluginCreator::getPluginName() const
{
    return INSTNORM3D_PLUGIN_NAME;
}

const char* InstanceNormalization3DPluginCreator::getPluginVersion() const
{
    return INSTNORM3D_PLUGIN_VERSION;
}

const PluginFieldCollection* InstanceNormalization3DPluginCreator::getFieldNames()
{
    return &mFC;
}

IPluginV2DynamicExt* InstanceNormalization3DPluginCreator::createPlugin(const char* name, const nvinfer1::PluginFieldCollection* fc)
{
    std::vector<float> scaleValues;
    std::vector<float> biasValues;
    float epsilon {};
    int relu {};
    float alpha {};
    const PluginField* fields = fc->fields;
    for (int i = 0; i < fc->nbFields; ++i)
    {
        const char* attrName = fields[i].name;
        if (!strcmp(attrName, "epsilon"))
        {
            ASSERT(fields[i].type == PluginFieldType::kFLOAT32);
            epsilon= *(static_cast<const float*>(fields[i].data));
        }
        else if (!strcmp(attrName, "scales"))
        {
            ASSERT(fields[i].type == PluginFieldType::kFLOAT32);
            int size = fields[i].length;
            scaleValues.reserve(size);
            const auto* w = static_cast<const float*>(fields[i].data);
            for (int j = 0; j < size; j++)
            {
                scaleValues.push_back(*w);
                w++;
            }
        }
        else if (!strcmp(attrName, "bias"))
        {
            ASSERT(fields[i].type == PluginFieldType::kFLOAT32);
            int size = fields[i].length;
            biasValues.reserve(size);
            const auto* w = static_cast<const float*>(fields[i].data);
            for (int j = 0; j < size; j++)
            {
                biasValues.push_back(*w);
                w++;
            }
        }
        else if (!strcmp(attrName, "relu"))
        {
            ASSERT(fields[i].type == PluginFieldType::kINT32);
            relu= *(static_cast<const int*>(fields[i].data));
        }
        else if (!strcmp(attrName, "alpha"))
        {
            ASSERT(fields[i].type == PluginFieldType::kFLOAT32);
            alpha= *(static_cast<const float*>(fields[i].data));
        }
    }

    Weights scaleWeights{DataType::kFLOAT, scaleValues.data(), (int64_t) scaleValues.size()};
    Weights biasWeights{DataType::kFLOAT, biasValues.data(), (int64_t) biasValues.size()};

    InstanceNormalization3DPlugin* obj = new InstanceNormalization3DPlugin(epsilon, scaleWeights, biasWeights, relu, alpha);
    obj->setPluginNamespace(mNamespace.c_str());
    obj->initialize();
    return obj;
}

IPluginV2DynamicExt* InstanceNormalization3DPluginCreator::deserializePlugin(const char* name, const void* serialData, size_t serialLength)
{
    InstanceNormalization3DPlugin* obj = new InstanceNormalization3DPlugin{serialData, serialLength}; 
    obj->setPluginNamespace(mNamespace.c_str());
    obj->initialize();
    return obj;
}
